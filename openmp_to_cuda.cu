#include "hip/hip_runtime.h"
// -mp=gpu -> Enable OpenMP targeting GPU and Multicore
// nvc++ -mp=gpu -acc openmp_to_cuda.cu
// nvprof --print-gpu-trace ./a.out
#include <iostream>
#include <omp.h>
#include <openacc.h>
#include <stdio.h>

__global__ void init(int*x) {
    int tid = threadIdx.x;
    x[tid] = 2;
}

int main() {

    int N = 10;
    int *x = new int[N];

    std::cout << "Number of devices : " << omp_get_num_devices() << std::endl;

    #pragma omp target data map(tofrom:x[0:N])
    {

        #pragma omp target data use_device_ptr(x)
        init<<<1,N>>>(x);

    }

    for (int i=0; i<N; ++i)
        std::cout << x[i] << " ";
    std::cout << std::endl;

    delete [] x;
    return 0;
}
